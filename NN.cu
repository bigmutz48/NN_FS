#include "hip/hip_runtime.h"
#include "NN.cuh"
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdexcept>


__host__ void NN::check_success_cudaMalloc(hipError_t cuda_call_return_value){} // end check_success_cudaMalloc()
__host__ bool NN::populate_nn_from_folder(std::string folder_path){} // end populate_nn_from_folder()
__device__ void NN::run_forward_prop_next_layer(){} // end run_forward_prop_next_layer()
__global__ void NN::run_test_one_example(float* inputs, int number_of_inputs){} // end run_test_one_example()


__global__ void NN::run_training_one_example(float* inputs, int number_of_inputs){

	// see if the network is setup for training 
	if(!this->is_ready_for_training){
		this-> set_up_nn_for_training();
	}


} // end run_training_one_example()
__host__ void NN::save_nn(std::string output_file_name){} // end save_nn


__host__ void NN::set_up_nn_for_training(){

	if(this->is_ready_for_training){
		throw std::runtime_error("NN::set_up_nn_for_training was called but is_ready_for_training was true");
	}

	//////// SETTING UP NN FOR TRAINING //////////
	
	// allocate memory for weights gradients
	hipError_t cudaCallReturnValue = hipMalloc((void** )&this->hidden_layers_weights_ptr_ptr_ptr,
			this->qty_hidden_layers * sizeof(float**));
	check_success_cudaMalloc(cudaCallReturnValue);

	for(int i = 0 ; i < this->qty_hidden_layers ; i++){
		
		
		hipError_t cudaCallReturnValue = hipMalloc((void** )&this->hidden_layers_weights_ptr_ptr_ptr[i],
				qty_neurons_per_hidden_layer * sizeof(float*));
		check_success_cudaMalloc(cudaCallReturnValue);


		}

		for(int j = 0 ; j < this->qty_neurons_per_hidden_layer ; j++){

			// special case for first layer
			if(i = 0){
			hipError_t cudaCallReturnValue = hipMalloc((void** )&this->hidden_layers_weights_ptr_ptr_ptr[i][j],
					this->qty_neurons_input_layer * sizeof(float));
			check_success_cudaMalloc(cudaCallReturnValue);

			}else{
			hipError_t cudaCallReturnValue = hipMalloc((void** )&this->hidden_layers_weights_ptr_ptr_ptr[i][j],
					this->qty_neurons_per_hidden_layer * sizeof(float));
			check_success_cudaMalloc(cudaCallReturnValue);
			}
		}

	}




	// once we have set up the nn then we need to set is_ready_for_training to true 
	this->is_ready_for_training = true;

} // end set_up_nn_for_training()



} // end nn_ready_to_setup

// 2nd helper for initialize_new_nn
__host__ void initialize_biases(){





} // end of initialize_biases


// helper for initialize_new_nn()
__host__ void initialize_weights(){



} // end initialize_weights_and_biases()


// use He intialization to initialize one layers of weights
__host__ void He_initialization_for_weights_one_layer(float* weights_ptr, int num_weights){

} // end He_initialization_for_weights_one_layer

__host__ void NN::initialize_new_nn(){

	

	// first allocate memory 
	this->allocate_memory_on_device();	


	// now we need to set all the weights to random values
	this->initialize_weights();
	
	// then initialize the biases
	this->initialize_biases();

} // end initialize_new_nn()

// when all the parameters are set for the NN dimensions we can allocate memory for them
__host__ void NN::allocate_memory_on_device(){

		//////////////// VALIDATE REQUEST ////////////////////////  
		
		if(this->qty_neurons_input_layer < 1){
throw std::runtime_error("NN::allocate_memory_on_device was called but qty_neurons_input_layer is less than 1");
		}

		if(this->qty_neurons_per_hidden_layer < 1){
throw std::runtime_error("NN::allocate_memory_on_device was called but qty_neurons_per_hidden_layer was less than 1");
		}

		if(this->qty_neurons_output_layer < 1){
throw std::runtime_error("NN::allocate_memory_on_device was called but qty_neurons_output_layer was less than zero");		
		}

		if(this->qty_hidden_layers < 1){
throw std::runtime_error("NN::allocate_memory_on_device was called but qty_hidden_layers was less than 1");
		}
		//////// END VALIDATE REQUEST //////////////////////


		//////////// ALLOCATE MEMORY ON GPU //////////////// 
		
		// NEURONS // 

		// set up input layer 
		hipError_t cudaMallocCall = hipMalloc((void** )&(this->input_layer_neurons_ptr), 
				this->qty_neurons_input_layer * sizeof(float));
		
		// make sure the allocation was succsessful
		this->check_success_cudaMalloc(cudaMallocCall);

		// set up the hidden layers main array 
		cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_neurons_ptr_ptr), 
				this->qty_hidden_layers * sizeof(float*));
		this->check_success_cudaMalloc(cudaMallocCall);

		// allocate space for each of the hidden layer arrays 
		for( int i = 0 ; i < qty_hidden_layers ; i++){
			
			// allocate space for the current hidden layer array 
			cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_neurons_ptr_ptr[i]), 
					this->qty_neurons_per_hidden_layer * sizeof(float));
			this->check_success_cudaMalloc(cudaMallocCall);
		}

		// set up output layer neurons 
		cudaMallocCall = hipMalloc((void** )&(this->output_layer_neurons_ptr),
				this->qty_neurons_output_layer * sizeof(float));
		check_success_cudaMalloc(cudaMallocCall);

		// WEIGHTS //  
		
		// allocate for the big array where each index represents the weights matrix for a single hidden layer
		cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_weights_ptr_ptr_ptr),
				this->qty_hidden_layers * sizeof(float**));
		check_success_cudaMalloc(cudaMallocCall);

		// allocate for each hidden layer 
		for(int i = 0 ; i < qty_hidden_layers ; i++){
			
			// if we are looking at the first hidden layer it is a special case 
			if(i == 0){
				cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_weights_ptr_ptr_ptr[i]),
						this->qty_neurons_per_hidden_layer * sizeof(float*));
				check_success_cudaMalloc(cudaMallocCall);
				
				// then go allocate memory for the weights vector of each neuron in the 
				// first hidden layer 
				for(int j = 0 ; j < this->qty_neurons_per_hidden_layer ; j++){
					
					// at each neuron allocate for a vector of floats the length of 
					// the number of input neurons 
					cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_weights_ptr_ptr_ptr[i][j]),
							this->qty_neurons_input_layer * sizeof(float));
					check_success_cudaMalloc(cudaMallocCall);
				}


			} else {

			// allocate memory for this layers main array 
			cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_weights_ptr_ptr_ptr[i]), 
							this->qty_neurons_per_hidden_layer * sizeof(float*));
			check_success_cudaMalloc(cudaMallocCall);

			// allocate memory for each vector in this hidden layers main array 
			for(int j = 0 ; j < this->qty_neurons_per_hidden_layer ; j++){
				cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_weights_ptr_ptr_ptr[i][j]), 
							this->qty_neurons_per_hidden_layer * sizeof(float));
				check_success_cudaMalloc(cudaMallocCall);


			}



		}
		} // end of hidden layer memory allocation for loop 

		// allocate for the output layer 
		cudaMallocCall = hipMalloc((void** )&(this->output_layer_weights_ptr_ptr),
				this->qty_neurons_per_hidden_layer * sizeof(float*));
		check_success_cudaMalloc(cudaMallocCall);

		// now allocate memory for the weights vector of each neuron in the output layer 
		for(int i = 0 ; i < this->qty_neurons_output_layer ; i++){

			cudaMallocCall = hipMalloc((void** )&(this->output_layer_weights_ptr_ptr[i]), 
					this->qty_neurons_per_hidden_layer * sizeof(float));
			check_success_cudaMalloc(cudaMallocCall);

		}



		// BIASES // 
		
		// allocate for hidden layers 
		cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_biases_ptr_ptr), // main array
				this->qty_hidden_layers * sizeof(float*));
		check_success_cudaMalloc(cudaMallocCall);
		for(int i = 0 ; i < this->qty_hidden_layers ; i++){ // array for each hidden layer 
			cudaMallocCall = hipMalloc((void** )&(this->hidden_layers_biases_ptr_ptr[i]),
					this->qty_neurons_per_hidden_layer * sizeof(float));
			check_success_cudaMalloc(cudaMallocCall);
		}

		// for output layer 
		cudaMallocCall = hipMalloc((void** )&(this->output_layer_biases_ptr), 
				this->qty_neurons_output_layer * sizeof(float));



} // end of allocate_memory_on_device()


